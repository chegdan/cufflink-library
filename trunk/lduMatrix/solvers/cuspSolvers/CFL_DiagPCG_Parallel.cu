/**********************************************************************\
  ______  __    __   _______  _______  __       __   __   __   __  ___     
 /      ||  |  |  | |   ____||   ____||  |     |  | |  \ |  | |  |/  /     
|  ,----'|  |  |  | |  |__   |  |__   |  |     |  | |   \|  | |  '  /  
|  |     |  |  |  | |   __|  |   __|  |  |     |  | |  . `  | |    <  
|  `----.|  `--'  | |  |     |  |     |  `----.|  | |  |\   | |  .  \
 \______| \______/  |__|     |__|     |_______||__| |__| \__| |__|\__\

Cuda For FOAM Link

cufflink is a library for linking numerical methods based on Nvidia's 
Compute Unified Device Architecture (CUDA™) C/C++ programming language
and OpenFOAM®.

Please note that cufflink is not approved or endorsed by OpenCFD® 
Limited, the owner of the OpenFOAM® and OpenCFD® trademarks and 
producer of OpenFOAM® software.

The official web-site of OpenCFD® Limited is www.openfoam.com .

------------------------------------------------------------------------
This file is part of cufflink.

    cufflink is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    cufflink is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with cufflink.  If not, see <http://www.gnu.org/licenses/>.    

    Author
        Daniel P. Combest.  All rights reserved.

    Description
        Algebraic Multigrid (AMG) based on Smoothed Aggregation
	preconditioned conjugate gradient solver for symmetric 
	matrices using a CUSP CUDA™ based solver.
                                                             
\**********************************************************************/

extern "C" void CFL_DiagPCG_Parallel(cusp_equation_system *CES,  OFSolverPerformance *OFSP, const cpuInterfaces *OFInterfaces)
{
	//convert the interface matrices from OpenFOAM to device memory interfaces.
	gpuInterfaces CFLInterfaces(OFInterfaces);

	ValueType small = 1e-20;// used to prevent floating point exception

	// Populate A
	#include "../CFL_Headers/fillCOOMatrix.H"

	#include "../CFL_Headers/setParallelGPUStorage.H"

	if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp) {
		std::cout<<"   Using Cusp_Diagonal preconditioner\n";
	}

 	cusp::precond::diagonal<ValueType, MemorySpace> M(A);

 	// Start the krylov solver
    assert(A.num_rows == A.num_cols);        // sanity check

    const size_t N = A.num_rows;

    // allocate workspace
    cusp::array1d<ValueType,MemorySpace> y(N);
    cusp::array1d<ValueType,MemorySpace> z(N);
    cusp::array1d<ValueType,MemorySpace> r(N);
    cusp::array1d<ValueType,MemorySpace> p(N);
    cusp::array1d<ValueType,MemorySpace> Ap(N);

	cusp::array1d< ValueType, hostMemorySpace > ph(p.size(),0);//used in ApjLoop.H
	cusp::array1d<ValueType,hostMemorySpace> *pjh;
	pjh = new cusp::array1d<ValueType,hostMemorySpace>[CFLInterfaces.nParInterfaces];
	for(int j = 0;j<CFLInterfaces.nParInterfaces;j++){ pjh[j] = cusp::array1d<ValueType,hostMemorySpace> (OFInterfaces->nColsInterface[j]);	}
        
    // y <- Ax
    cusp::multiply(A, X, y);

    //take into account the interfaces
    #include "AXLoop.H"

    //define the normalization factor
    ValueType normFactor = 1.0;

    #include "../CFL_Headers/buildGlobalNormFactor.H"

    // r <- b - A*x
    cusp::blas::axpby(B, y, r, ValueType(1), ValueType(-1));
   
    // z <- M*r
    cusp::multiply(M, r, z);

    // p <- z
    cusp::blas::copy(z, p);
		
    // rz = <r^H, z>
    ValueType rz = gpuSumProd(r, z);

    ValueType normR = gpuSumMag(r)/normFactor;
    ValueType normR0 = normR;//initial residual
    OFSP->iRes	= normR0;
    int count = 0;

 	if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp){std::cout << "   Iteration "<<count<<" residual = "<< std::setw(10) << normR << std::endl;}
	
    while ( normR > (OFSP->tol) && count<= (OFSP->maxIter) && normR/normR0 >= (OFSP->relTol))
    {
        // Ap <- A*p
        cusp::multiply(A, p, Ap);

		//take into account the interfaces
		#include "ApjLoop.H"
        
        // alpha <- <r,z>/<Ap,p>
        ValueType alpha =  rz / gpuSumProd(Ap, p);

        // x <- x + alpha * p
        cusp::blas::axpy(p, X, alpha);

        // r <- r - alpha * Ap		
        cusp::blas::axpy(Ap, r, -alpha);

        // z <- M*r
        cusp::multiply(M, r, z);
		
        ValueType rz_old = rz;

        // rz = <r^H, z>
        rz = gpuSumProd(r, z);

        // beta <- <r_{i+1},r_{i+1}>/<r,r> 
        ValueType beta = rz / rz_old;
		
        // p <- r + beta*p should be p <- z + beta*p
        cusp::blas::axpby(z, p, p, ValueType(1), beta);
	
		normR = gpuSumMag(r)/normFactor;;
	
		count++;

		if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp) {
			std::cout << "   Iteration "<<count<<" residual = "<< std::setw(10) << normR << std::endl;
		}
    } //end the krylov solver

	//final residual
	OFSP->fRes = normR;
	OFSP->nIterations = count;

	//converged?
	if(OFSP->fRes<=OFSP->tol || OFSP->fRes/OFSP->iRes<=OFSP->relTol)
		OFSP->converged=true;
	else
		OFSP->converged=false;

	//pass the solution vector back	
	CES->X = X;
}
