/**********************************************************************\
  ______  __    __   _______  _______  __       __   __   __   __  ___     
 /      ||  |  |  | |   ____||   ____||  |     |  | |  \ |  | |  |/  /     
|  ,----'|  |  |  | |  |__   |  |__   |  |     |  | |   \|  | |  '  /  
|  |     |  |  |  | |   __|  |   __|  |  |     |  | |  . `  | |    <   
|  `----.|  `--'  | |  |     |  |     |  `----.|  | |  |\   | |  .  \  
 \______| \______/  |__|     |__|     |_______||__| |__| \__| |__|\__\ 

Cuda For FOAM Link

cufflink is a library for linking numerical methods based on Nvidia's 
Compute Unified Device Architecture (CUDA™) C/C++ programming language
and OpenFOAM®.

Please note that cufflink is not approved or endorsed by OpenCFD® 
Limited, the owner of the OpenFOAM® and OpenCFD® trademarks and 
producer of OpenFOAM® software.

The official web-site of OpenCFD® Limited is www.openfoam.com .

------------------------------------------------------------------------
This file is part of cufflink.

    cufflink is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    cufflink is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with cufflink.  If not, see <http://www.gnu.org/licenses/>.    

    Author
        Daniel P. Combest.  All rights reserved.

    Description
        Sparse approximate inverse preconditioned conjugate gradient stabilized 
	solver for asymmetric Matrices using a CUSP CUDA™ based solver.
                                                             
\**********************************************************************/

extern "C" void CFL_AinvPBiCGStab_Parallel(cusp_equation_system *CES,  OFSolverPerformance *OFSP, const cpuInterfaces *OFInterfaces){

	//convert the interface matrices from OpenFOAM to device memory interfaces.
	gpuInterfaces CFLInterfaces(OFInterfaces);

	ValueType small = 1e-20;// used to prevent floating point exception

	cusp::coo_matrix<IndexType, ValueType, MemorySpace> A(CES->A);
	cusp::array1d<ValueType, MemorySpace> X(CES->X);
	cusp::array1d<ValueType, MemorySpace> B(CES->B);

	//fill in the rest of the diag (rows and col)
	thrust::sequence(A.row_indices.begin(),A.row_indices.begin()+OFSP->nCells);//determine row indices of diagonal values and fill A COO matrix
	thrust::sequence(A.column_indices.begin(),A.column_indices.begin()+OFSP->nCells);//determine column indices of diagonal values and fill A COO matrix

	A.sort_by_row_and_column();//sorted coo by row and column. speeds code up a little bit more

	//choose and convert to correct matrix format	
	#include "../CFL_Headers/setParallelGPUStorage.H"

	if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp){std::cout<<"   Using Cusp_Ainv preconditioner - non-symmetric Bridson Ainv with novel dropping strategy and linStrategy = "<< OFSP->linStrategy<<"\n";}

	cusp::precond::bridson_ainv<ValueType, MemorySpace> M(A, 0, -1, true, int(OFSP->linStrategy));

//start Krylov solver
    assert(A.num_rows == A.num_cols);        // sanity check

    const size_t N = A.num_rows;

    // allocate workspace
    cusp::array1d<ValueType,MemorySpace> y(N);

    cusp::array1d<ValueType,MemorySpace>   p(N);
    cusp::array1d<ValueType,MemorySpace>   r(N);
    cusp::array1d<ValueType,MemorySpace>   r_star(N);
    cusp::array1d<ValueType,MemorySpace>   s(N);
    cusp::array1d<ValueType,MemorySpace>  Mp(N);
    cusp::array1d<ValueType,MemorySpace> AMp(N);
    cusp::array1d<ValueType,MemorySpace>  Ms(N);
    cusp::array1d<ValueType,MemorySpace> AMs(N);

	cusp::array1d< ValueType, hostMemorySpace > Msh(Ms.size(),0);
	cusp::array1d< ValueType, hostMemorySpace > Mph(Mp.size(),0);
	cusp::array1d< ValueType, hostMemorySpace > *Mpjh;
	cusp::array1d< ValueType, hostMemorySpace > *Msjh;
	Mpjh = new cusp::array1d<ValueType,hostMemorySpace>[CFLInterfaces.nParInterfaces];
	Msjh = new cusp::array1d<ValueType,hostMemorySpace>[CFLInterfaces.nParInterfaces];

	for(int j = 0;j<CFLInterfaces.nParInterfaces;j++){ 
		Mpjh[j] = cusp::array1d<ValueType,hostMemorySpace> (OFInterfaces->nColsInterface[j]);	
		Msjh[j] = cusp::array1d<ValueType,hostMemorySpace> (OFInterfaces->nColsInterface[j]);	
	}


 // y <- Ax
    cusp::multiply(A, X, y);

    #include "AXLoop.H"

    //define the normalization factor
    ValueType normFactor = 1.0;

    #include "../CFL_Headers/buildGlobalNormFactor.H"

    // r <- b - A*x
    cusp::blas::axpby(B, y, r, ValueType(1), ValueType(-1));

    // p <- r
    cusp::blas::copy(r, p);

    // r_star <- r
    cusp::blas::copy(r, r_star);

    ValueType r_r_star_old = gpuSumProd(r_star, r);


    ValueType normR = gpuSumMag(r)/normFactor;
    ValueType normR0 = normR;//initial residual
    OFSP->iRes	= normR0;
    int count = 0;

 	if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp){std::cout << "   Iteration "<<count<<" residual = "<< std::setw(10) << normR << std::endl;}
	
    while ( normR > (OFSP->tol) && count<= (OFSP->maxIter) && normR/normR0 >= (OFSP->relTol))
    {
        // Mp = M*p
        cusp::multiply(M, p, Mp);

        // AMp = A*Mp
        cusp::multiply(A, Mp, AMp);

    	#include "AMpLoop.H"

        // alpha = (r_j, r_star) / (A*M*p, r_star)
        ValueType alpha = r_r_star_old / gpuSumProd(r_star, AMp);
        
        // s_j = r_j - alpha * AMp
        cusp::blas::axpby(r, AMp, s, ValueType(1), ValueType(-alpha));

	ValueType normS = gpuSumMag(s)/normFactor;

	if (!( normS > (OFSP->tol) && normS/normR0 >= (OFSP->relTol))){//is this right?
	  // x += alpha*M*p_j
	  cusp::blas::axpby(X, Mp, X, ValueType(1), ValueType(alpha));
          
          // y <- Ax
          cusp::multiply(A, X, y);

	  #include "AX2Loop.H"

          // r <- b - A*x
          cusp::blas::axpby(B, y, r, ValueType(1), ValueType(-1));

	  normR = gpuSumMag(r)/normFactor;

	  count++;

	  if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp){std::cout << "   Iteration "<<count<<" residual = "<< std::setw(10) << normR << std::endl;}
	  break;
	}

        // Ms = M*s_j
        cusp::multiply(M, s, Ms);
        
        // AMs = A*Ms
        cusp::multiply(A, Ms, AMs);

    	#include "AMsLoop.H"

        // omega = (AMs, s) / (AMs, AMs)
        ValueType omega = gpuSumProd(AMs, s) / gpuSumProd(AMs, AMs);
        
        // x_{j+1} = x_j + alpha*M*p_j + omega*M*s_j
        cusp::blas::axpbypcz(X, Mp, Ms, X, ValueType(1), alpha, omega);

        // r_{j+1} = s_j - omega*A*M*s
        cusp::blas::axpby(s, AMs, r, ValueType(1), -omega);

        // beta_j = (r_{j+1}, r_star) / (r_j, r_star) * (alpha/omega)
        ValueType r_r_star_new = gpuSumProd(r_star, r);
        ValueType beta = (r_r_star_new / r_r_star_old) * (alpha / omega);
        r_r_star_old = r_r_star_new;

        // p_{j+1} = r_{j+1} + beta*(p_j - omega*A*M*p)
        cusp::blas::axpbypcz(r, p, AMp, p, ValueType(1), beta, -beta*omega);

	normR = gpuSumMag(r)/normFactor;

	count++;

	if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp){std::cout << "   Iteration "<<count<<" residual = "<< std::setw(10) << normR << std::endl;}
    }
//end Krylov Solver

	//final residual
	OFSP->fRes = gpuSumMag(r)/normFactor;
	OFSP->nIterations = count;

	//converged?
	if(OFSP->fRes<=OFSP->tol || OFSP->fRes/OFSP->iRes<=OFSP->relTol)
		OFSP->converged=true;
	else
		OFSP->converged=false;

	//pass the solution vector back	
	CES->X = X;


}
