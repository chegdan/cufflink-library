/**********************************************************************\
  ______  __    __   _______  _______  __       __   __   __   __  ___     
 /      ||  |  |  | |   ____||   ____||  |     |  | |  \ |  | |  |/  /     
|  ,----'|  |  |  | |  |__   |  |__   |  |     |  | |   \|  | |  '  /  
|  |     |  |  |  | |   __|  |   __|  |  |     |  | |  . `  | |    <   
|  `----.|  `--'  | |  |     |  |     |  `----.|  | |  |\   | |  .  \
 \______| \______/  |__|     |__|     |_______||__| |__| \__| |__|\__\

Cuda For FOAM Link

cufflink is a library for linking numerical methods based on Nvidia's 
Compute Unified Device Architecture (CUDA™) C/C++ programming language
and OpenFOAM®.

Please note that cufflink is not approved or endorsed by OpenCFD® 
Limited, the owner of the OpenFOAM® and OpenCFD® trademarks and 
producer of OpenFOAM® software.

The official web-site of OpenCFD® Limited is www.openfoam.com .

------------------------------------------------------------------------
This file is part of cufflink.

    cufflink is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    cufflink is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with cufflink.  If not, see <http://www.gnu.org/licenses/>.    

    Author
        Daniel P. Combest.  All rights reserved.

    Description
        diagonal preconditioned conjugate gradient 
	solver for symmetric Matrices using a CUSP CUDA™ based solver.
                                                             
\**********************************************************************/

extern "C" void CFL_CG_Parallel(cusp_equation_system *CES,  OFSolverPerformance *OFSP, const cpuInterfaces *OFInterfaces)
{
	//convert the interface matrices from OpenFOAM to device memory interfaces.
	gpuInterfaces CFLInterfaces(OFInterfaces);

	ValueType small = 1e-20;// used to prevent floating point exception

	// Populate A
	#include "../CFL_Headers/fillCOOMatrix.H"
	
	#include "../CFL_Headers/setParallelGPUStorage.H"

	const size_t N = A.num_rows;

	// allocate workspace
	cusp::array1d<ValueType,MemorySpace> y(N,0);
	cusp::array1d<ValueType,MemorySpace> r(N,0);

    	// y <- Ax
	cusp::multiply(A, X, y);

	#include "AXLoop.H"

	// r <- b - A*x
	cusp::blas::axpby(B, y, r, ValueType(1), ValueType(-1));

	ValueType normFactor = 1.0;
	
	#include "../CFL_Headers/buildGlobalNormFactor.H"

	//start the krylov solver

	assert(A.num_rows == A.num_cols);        // sanity check
	
	// allocate workspace
	cusp::array1d<ValueType,MemorySpace> Ap(N);
	cusp::array1d<ValueType,MemorySpace> rold(N);
	cusp::array1d<ValueType,MemorySpace> p(N);

	cusp::array1d< ValueType, hostMemorySpace > ph(p.size(),0);//used in ApjLoop.H
	cusp::array1d<ValueType,hostMemorySpace> *pjh;
	pjh = new cusp::array1d<ValueType,hostMemorySpace>[CFLInterfaces.nParInterfaces];
	for(int j = 0;j<CFLInterfaces.nParInterfaces;j++){ pjh[j] = cusp::array1d<ValueType,hostMemorySpace> (OFInterfaces->nColsInterface[j]);	}

	//cusp::array1d< ValueType, hostMemorySpace > pjh(CFLInterfaces.nColsInterface[j],0);



	ValueType alpha = 1;
	ValueType beta = 1;
	        
  	// r0 -> p0
  	cusp::blas::copy(r, p);

    ValueType normR = gpuSumMag(r)/normFactor;
    ValueType normR0 = normR;//initial residual
    OFSP->iRes	= normR0;
    int count = 0;

 	if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp){
 		std::cout << "   Iteration "<<count<<" residual = "<< std::setw(10) << normR << std::endl;
 	}
	
    while ( normR > (OFSP->tol) && count<= (OFSP->maxIter) && normR/normR0 >= (OFSP->relTol))
    {
        // Apj <- A*p
        cusp::multiply(A, p, Ap);

		#include "ApjLoop.H"
      
        // alpha <- <r,z>/<Ap,p>
    	alpha =  gpuSumProd(r,r) / gpuSumProd(Ap, p);

        // x <- x + alpha * p
        cusp::blas::axpy(p, X, alpha);

        //copy rold<-r
        cusp::blas::copy(r, rold);

        // r <- r - alpha * y		
        cusp::blas::axpy(Ap, r, -alpha);

        // beta <- <r,r>/<r_old,r_old> 
        beta = gpuSumProd(r, r) / gpuSumProd(rold, rold);
		
        // p <- r + beta*p should be p <- z + beta*p
        cusp::blas::axpby(r, p, p, ValueType(1), beta);

        normR = gpuSumMag(r)/normFactor;

        count++;

        if(0 == CFLInterfaces.myThreadNumber && OFSP->debugCusp){
        	std::cout << "   Iteration "<<count<<" residual = "<< std::setw(10) << normR << std::endl;
        }
    }

 	//end the krylov solver

	//final residual
	OFSP->fRes = normR;
	OFSP->nIterations=count;

	//converged?
	if(OFSP->fRes<=OFSP->tol || OFSP->fRes/OFSP->iRes<=OFSP->relTol)
		OFSP->converged=true;
	else
		OFSP->converged=false;

	//pass the solution vector back	
	CES->X = X;
}
