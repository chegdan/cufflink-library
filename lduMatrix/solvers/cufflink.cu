/**********************************************************************\
  ______  __    __   _______  _______  __       __   __   __   __  ___     
 /      ||  |  |  | |   ____||   ____||  |     |  | |  \ |  | |  |/  /     
|  ,----'|  |  |  | |  |__   |  |__   |  |     |  | |   \|  | |  '  /  
|  |     |  |  |  | |   __|  |   __|  |  |     |  | |  . `  | |    <   
|  `----.|  `--'  | |  |     |  |     |  `----.|  | |  |\   | |  .  \
 \______| \______/  |__|     |__|     |_______||__| |__| \__| |__|\__\

Cuda For FOAM Link

cufflink is a library for linking numerical methods based on Nvidia's 
Compute Unified Device Architecture (CUDA™) C/C++ programming language
and OpenFOAM®.

Please note that cufflink is not approved or endorsed by OpenCFD® 
Limited, the owner of the OpenFOAM® and OpenCFD® trademarks and 
producer of OpenFOAM® software.

The official web-site of OpenCFD® Limited is www.openfoam.com .

------------------------------------------------------------------------
This file is part of cufflink.

    cufflink is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    cufflink is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with cufflink.  If not, see <http://www.gnu.org/licenses/>.    

    Author
        Daniel P. Combest.  All rights reserved.

    Description
        A header file to compile the extern "C" functions containing the 
	CUSP CUDA™ based solvers.  This avoids problems with the multiple 
	definitions of some functions.
                                                             
\**********************************************************************/
#include "hip/hip_runtime.h"

//System Includes
#include <mpi.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <stdio.h>
#include <limits>

//CUSP Includes
#include <cusp/detail/config.h>
#include <cusp/verify.h>
#include <cusp/precond/ainv.h>
#include <cusp/precond/diagonal.h>
#include <cusp/precond/smoothed_aggregation.h>
#include <cusp/csr_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/dia_matrix.h>
#include <cusp/blas.h>
#include <cusp/multiply.h>

//THRUST Includes
#include <thrust/reduce.h>//for the summation of components of a vector
#include <thrust/sequence.h>//for diagonal construction
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h> 

//CUFFLINK Includes
#include "CFL_Headers/cuspTypeDefs.H"//change this file ( along with passing the arch flag to sm_10 in nvccwmake script ) if you want single precision

#include "CFL_Headers/OFSolverPerformance.H"
#include "CFL_Headers/cusp_equation_system.H"
#include "CFL_Headers/cpuInterfaces.C"
#include "CFL_Headers/gpuInterfaces.C"
#include "CFL_Headers/globalOps.H"

//CUFFLINK extern function definitions
#include "cuspSolvers/CFL_AinvPCG.cu"
#include "cuspSolvers/CFL_AinvPCG_Parallel.cu"

#include "cuspSolvers/CFL_CG.cu"
#include "cuspSolvers/CFL_CG_Parallel.cu"

#include "cuspSolvers/CFL_DiagPCG.cu"
#include "cuspSolvers/CFL_DiagPCG_Parallel.cu"

#include "cuspSolvers/CFL_SmAPCG.cu"
#include "cuspSolvers/CFL_SmAPCG_Parallel.cu"

#include "cuspSolvers/CFL_DiagPBiCGStab.cu"
#include "cuspSolvers/CFL_DiagPBiCGStab_Parallel.cu"

#include "cuspSolvers/CFL_AinvPBiCGStab.cu"
#include "cuspSolvers/CFL_AinvPBiCGStab_Parallel.cu"


